
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void random_ints(int* a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
        a[i] = rand();
}

# define N (2048*2048)
# define THREADS_PER_BLOCK 512
int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // cp inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // launch add kernel on GPU with N threads
    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

    // cp results to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    scanf("%d\n", d_c);

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
