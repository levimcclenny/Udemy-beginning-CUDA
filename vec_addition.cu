#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    c[clockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
# define N 512
int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);


    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // cp inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // launch add kernel on GPU with N blocks
    add<<<N,1>>>(d_a, d_b, d_c);

    // cp results to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("%i\n", c);

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
